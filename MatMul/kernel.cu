#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <locale>
#include <windows.h>
#include <string>

using namespace std;

void print_matrix(double* matrix, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            cout << matrix[i + j] << " ";
        }
        cout << endl;
    }
}

void init_matrix(double* matrix, int n) {
    for (int i = 0; i < n * n; i++) {
        matrix[i] = rand() / 100;
    }
}

__global__ void mul_matrix_kernel(double* A, double* B, double* C, int n) {
    // Реализация перемножения матриц на GPU CUDA
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    C[n * row + col] = 0;

    if (row < n && col < n) {
        for (int k = 0; k < n; k++) {
            C[n * row + col] += A[row * n + k] * B[k * n + col];
        }
    }
}

void mul_matrix_gpu(double* A, double* B, double* C, int n, float time) {
    dim3 threadsPerBlock(n, n);
    dim3 blocksPerGrid(1, 1);
    if (n * n > 512) {
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
        blocksPerGrid.x = ceil(double(n) / double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(n) / double(threadsPerBlock.y));
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    mul_matrix_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, n);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void mul_matrix_cpu(double* A, double* B, double* C, int n) {
    // Реализация перемножения матриц на CPU
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            double sum = 0;
            for (int k = 0; k < n; k++) {
                sum += A[i * n + k] * B[k * n + j];
            }
            C[i * n + j] = sum;
        }
    }
}

bool equals_matrix(double* A, double* B, int n) {
    for (int i = 0; i < n * n; i++) {
        if (A[i] != B[i]) {
            return false;
        }
    }
    return true;
}




int main()
{
    setlocale(LC_ALL, "Russian");

    srand(time(nullptr));

    cout << "Введите количсетво строк и столбцов в матрице: ";
    int n;
    cin >> n;

    size_t bytes = n * n * sizeof(double);

    float time_cpu = 0;
    float time_gpu = 0;

    double *h_A, *h_B, *h_C_cpu, *h_C_gpu;

    hipHostMalloc((void**) &h_A, bytes);
    hipHostMalloc((void**) &h_B, bytes);
    hipHostMalloc((void**) &h_C_gpu, bytes);
    hipHostMalloc((void**) &h_C_cpu, bytes);

    init_matrix(h_A, n);
    init_matrix(h_B, n);

    double* d_A, * d_B, * d_C;

    hipMalloc((void**)&d_A, bytes);
    hipMalloc((void**)&d_B, bytes);
    hipMalloc((void**)&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyDeviceToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyDeviceToDevice);

    mul_matrix_gpu(d_A, d_B, d_C, n, time_gpu);

    hipMemcpy(h_C_gpu, d_C, bytes, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    time_cpu = clock();
    mul_matrix_cpu(h_A, h_B, h_C_cpu, n);
    time_cpu = clock() - time_cpu;

    if (n < 10) {
        cout << "Результат перемножения матрицы на GPU: " << endl;
        print_matrix(h_C_gpu, n);
        cout << "Результат перемножения матрицы на CPU: " << endl;
        print_matrix(h_C_cpu, n);
    }
    else {
        cout << "Результирующие матрицы на GPU и CPU равны? " << equals_matrix(h_C_cpu, h_C_gpu, n) << endl;
    }

    cout << "Время работы программы на GPU (мc): " << time_gpu << endl;
    cout << "Время работы программы на CPU (мc): " << time_cpu << endl;
    cout << "Ускорение " << time_cpu / time_gpu << endl;

    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C_gpu);
    hipHostFree(h_C_cpu);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    system("pause");
    return 0;
}

